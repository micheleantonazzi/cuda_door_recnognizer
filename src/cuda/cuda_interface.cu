#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "cuda_interface.h"
#include "utilities/gpu_utilities.h"
#include "../utilities/time_utilities.h"

__global__ void test_kernel(){
    printf("Hello from\n");
}

void CudaInterface::test_cuda(){
    test_kernel<<<1, 10>>>();
    CHECK(hipDeviceSynchronize());
}

__global__ void to_gray_scale(unsigned char *destination, unsigned char *source, int width, int height){
    int threadTot = gridDim.x * blockDim.x;

    int imageSize = width * height * 3;

    int valuesPerThread = (imageSize / threadTot) + 3;

    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId * valuesPerThread < imageSize){

        // Move the pointer to the correct position
        source += threadId * valuesPerThread;
        destination += threadId * valuesPerThread;

        int start = threadId * valuesPerThread;

        for(int i = 0; i < valuesPerThread && start + i < imageSize; i += 3){
            unsigned char average = (*(source++) + *(source++) + *(source++)) / 3;
            *(destination++) = average;
            *(destination++) = average;
            *(destination++) = average;

        }
    }
}

double CudaInterface::toGrayScale(unsigned char *destination, unsigned char *source, int width, int height, int numBlocks, int numThread) {

    double time = seconds();

    to_gray_scale<<<numBlocks, numThread>>>(destination, source, width, height);

    hipDeviceSynchronize();

    time = seconds() - time;

    return time;
}