#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "cuda_interface.h"
#include "utilities/gpu_utilities.h"
#include "../utilities/time_utilities.h"

__global__ void test_kernel(){
    printf("Hello from\n");
}

void CudaInterface::test_cuda(){
    test_kernel<<<1, 10>>>();
    CHECK(hipDeviceSynchronize());
}

Pixel24* CudaInterface::getPixelArray(unsigned char *imageData, int width, int height) {
    int imageSize = width * height;

    Pixel24 *pixelArray;

    hipHostMalloc(&pixelArray, imageSize * sizeof(Pixel24));

    for(int i = 0; i < imageSize; ++i){
        pixelArray[i].R = *(imageData++);
        pixelArray[i].G = *(imageData++);
        pixelArray[i].B = *(imageData++);
    }

    return pixelArray;

}

void CudaInterface::pixelArrayToCharArray(unsigned char *imageData, Pixel24 *source, int width, int height) {
    int imageSize = width * height;

    for (int i = 0; i < imageSize; ++i) {
        Pixel24 pixel = *(source++);
        *(imageData++) = pixel.R;
        *(imageData++) = pixel.G;
        *(imageData++) = pixel.B;
    }
}

__global__ void to_gray_scale(unsigned char *destination, unsigned char *source, int width, int height){
    int threadTot = gridDim.x * blockDim.x;

    int imageSize = width * height * 3;

    int valuesPerThread = (imageSize / threadTot) + 3;

    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId * valuesPerThread < imageSize){

        // Move the pointer to the correct position
        source += threadId * valuesPerThread;
        destination += threadId * valuesPerThread;

        int start = threadId * valuesPerThread;

        for(int i = 0; i < valuesPerThread && start + i < imageSize; i += 3){
            unsigned char average = (*(source++) + *(source++) + *(source++)) / 3;
            *(destination++) = average;
            *(destination++) = average;
            *(destination++) = average;

        }
    }
}

double CudaInterface::toGrayScale(unsigned char *destination, unsigned char *source, int width, int height, int numBlocks, int numThread) {

    double time = seconds();

    to_gray_scale<<<numBlocks, numThread>>>(destination, source, width, height);

    hipDeviceSynchronize();

    time = seconds() - time;

    return time;
}

__global__ void to_gray_scale(Pixel24 *destination, Pixel24 *source, int width, int height){

    int totThread = gridDim.x * blockDim.x;

    // Thread group is 32 (the warp dimension) if the total number of thread is equal or higher than warp dimension (32)
    int threadGroupDim = totThread >= 32 ? 32 : totThread;

    int imageSize = width * height;

    int jumpPerThreadGroup = (imageSize / totThread) + 1;

    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    // The group of a thread
    int threadGroup = threadId / threadGroupDim;

    // The number if thread inside his group
    int threadIdInGroup = (blockDim.x * blockIdx.x + threadIdx.x) % threadGroupDim;

    if (threadGroupDim * threadGroup * jumpPerThreadGroup + threadIdInGroup < imageSize){

        // Move the pointer to the correct position
        // In this way the accesses to global memory are aligned and coalescent
        source += threadGroup * threadGroupDim * jumpPerThreadGroup + threadIdInGroup;
        destination += threadGroup * threadGroupDim * jumpPerThreadGroup + threadIdInGroup;

        int start = threadGroup * jumpPerThreadGroup * threadGroupDim + threadIdInGroup;

        for(int i = 0; i < jumpPerThreadGroup && start + i * threadGroupDim < imageSize; i++){

            source += threadGroupDim;
            destination += threadGroupDim;

            Pixel24 pixel24 = *source;
            unsigned char average = (pixel24.R + pixel24.G + pixel24.B) / 3;
            pixel24.R = average;
            pixel24.G = average;
            pixel24.B = average;
            *destination = pixel24;
        }
    }
}

double CudaInterface::toGrayScale(Pixel24 *destination, Pixel24 *source, int width, int height, int numBlocks, int numThread) {
    double time = seconds();

    to_gray_scale<<<numBlocks, numThread>>>(destination, source, width, height);

    hipDeviceSynchronize();

    time = seconds() - time;

    return time;
}