#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_interface.h"
#include "utilities/gpu_utilities.h"
#include "../utilities/time_utilities.h"

__global__ void test_kernel(){
    printf("Hello from\n");
}

void CudaInterface::test_cuda(){
    test_kernel<<<1, 10>>>();
    CHECK(hipDeviceSynchronize());
}

__global__ void to_gray_scale(unsigned char *destination, unsigned char *source, int width, int height){
    printf("Hello from \n");
}

double CudaInterface::toGrayScale(unsigned char *destination, unsigned char *source, int width, int height, int numBlocks, int numThread) {

    int sizeImage = width * height * 3;

    unsigned char *sourceGpu;
    unsigned char *destinationGpu;

    hipMalloc(&sourceGpu, sizeof(unsigned char) * sizeImage);
    hipMalloc(&destinationGpu, sizeof(unsigned char) * sizeImage);

    hipMemcpy(sourceGpu, source, sizeImage, hipMemcpyHostToDevice);

    double time = seconds();

    to_gray_scale<<<numBlocks, numThread>>>(destinationGpu, sourceGpu, width, height);

    time = seconds() - time;

    hipMemcpy(destination, destinationGpu, sizeImage, hipMemcpyDeviceToHost);

    hipFree(sourceGpu);
    hipFree(destinationGpu);

    return time;
}