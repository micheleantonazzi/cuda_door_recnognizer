#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_interface.h"
#include "utilities/gpu_utilities.h"

__global__ void test_kernel(){
    printf("Hello from GPU\n");
}

void CudaInterface::test_cuda(){
    test_kernel<<<1, 10>>>();
    CHECK(hipDeviceSynchronize());
}