#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void test_kernel(){
    printf("Hello form gpu");
}

void test_cuda(){
    test_kernel<<<1, 10>>>();
    hipDeviceReset();
}